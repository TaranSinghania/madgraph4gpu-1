#include "hip/hip_runtime.h"
//==========================================================================
// This file has been automatically generated for C++ Standalone by
// MadGraph5_aMC@NLO v. 2.7.3.py3, 2020-06-28
// By the MadGraph5_aMC@NLO Development Team
// Visit launchpad.net/madgraph5 and amcatnlo.web.cern.ch
//==========================================================================

#include "CPPProcess.h"
#include "HelAmps_sm.h"

#include <algorithm> 
#include <iostream> 
#include <thrust/complex.h> 

using namespace MG5_sm; 

//==========================================================================
// Class member functions for calculating the matrix elements for
// Process: g g > t t~ WEIGHTED<=2 @1

__constant__ int cHel[16][4]; 
// __constant__ double cmME[4]; value hardcoded now
// extern __constant__ int cPerm[4];
// 
__constant__ double cIPC[4];  // coupling ?
__constant__ double cIPD[2]; 


// Evaluate |M|^2 for each subprocess

__device__ void calculate_wavefunctions(int ihel, char * dps, size_t dpt,
    double &matrix)
{
  thrust::complex<double> amp[3]; 
  // Calculate wavefunctions for all processes
  thrust::complex<double> w[5][6]; 
  vxxxxx((double * )(dps + 0 * dpt), 0., cHel[ihel][0], -1, w[0]); 
  vxxxxx((double * )(dps + 1 * dpt), 0., cHel[ihel][1], -1, w[1]); 
  oxxxxx((double * )(dps + 2 * dpt), cIPD[0], cHel[ihel][2], +1, w[2]); 
  ixxxxx((double * )(dps + 3 * dpt), cIPD[0], cHel[ihel][3], -1, w[3]); 
  VVV1P0_1(w[0], w[1], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[4]);
  // Amplitude(s) for diagram number 1
  FFV1_0(w[3], w[2], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[0]); //printf(" &amp[0] %f %f\n", amp[0].real(), amp[0].imag());
  FFV1_1(w[2], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[4]);
  // Amplitude(s) for diagram number 2
  FFV1_0(w[3], w[4], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[1]); //printf(" &amp[1] %f %f\n", amp[1].real(), amp[1].imag());
  FFV1_2(w[3], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[4]);
  // Amplitude(s) for diagram number 3
  FFV1_0(w[4], w[2], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[2]); //printf(" &amp[2] %f %f\n", amp[2].real(), amp[2].imag());
  // double CPPProcess::matrix_1_gg_ttx() {
  int i, j; 
  // Local variables

  // const int ngraphs = 2;
  const int ncolor = 2; 
  thrust::complex<double> ztemp; 
  thrust::complex<double> jamp[ncolor]; 
  // The color matrix;
  static const double denom[ncolor] = {3, 3}; 
  static const double cf[ncolor][ncolor] = {{16, -2}, {-2, 16}}; 

  // Calculate color flows
  jamp[0] = +thrust::complex<double> (0, 1) * amp[0] - amp[1]; 
  jamp[1] = -thrust::complex<double> (0, 1) * amp[0] - amp[2]; 

  // Sum and square the color flows to get the matrix element
  for(i = 0; i < ncolor; i++ )
  {
    ztemp = 0.; 
    for(j = 0; j < ncolor; j++ )
      ztemp = ztemp + cf[i][j] * jamp[j]; 
    matrix = matrix + (ztemp * conj(jamp[i])).real()/denom[i]; 
  }

  // Store the leading color flows for choice of color
  // for(i=0;i < ncolor; i++)
  // jamp2[0][i] += real(jamp[i]*conj(jamp[i]));

}



CPPProcess::CPPProcess(int numiterations, int gpublocks, int gputhreads, 
bool verbose, bool debug)
: m_numiterations(numiterations), gpu_nblocks(gpublocks), 
gpu_nthreads(gputhreads), dim(gpu_nblocks * gpu_nthreads) 
{


  // Helicities for the process - nodim
  static const int tHel[ncomb][nexternal] = {{-1, -1, -1, -1}, {-1, -1, -1, 1},
      {-1, -1, 1, -1}, {-1, -1, 1, 1}, {-1, 1, -1, -1}, {-1, 1, -1, 1}, {-1, 1,
      1, -1}, {-1, 1, 1, 1}, {1, -1, -1, -1}, {1, -1, -1, 1}, {1, -1, 1, -1},
      {1, -1, 1, 1}, {1, 1, -1, -1}, {1, 1, -1, 1}, {1, 1, 1, -1}, {1, 1, 1,
      1}};
  hipMemcpyToSymbol(HIP_SYMBOL(cHel), tHel, ncomb * nexternal * sizeof(int)); 
  // perm - nodim
  // static int perm[nexternal] = {0, 1, 2, 3};
}

CPPProcess::~CPPProcess() {}

const std::vector<double> &CPPProcess::getMasses() const {return mME;}

//--------------------------------------------------------------------------
// Initialize process.

void CPPProcess::initProc(string param_card_name) 
{
  // Instantiate the model class and set parameters that stay fixed during run
  pars = Parameters_sm::getInstance(); 
  SLHAReader slha(param_card_name); 
  pars->setIndependentParameters(slha); 
  pars->setIndependentCouplings(); 
  pars->printIndependentParameters(); 
  pars->printIndependentCouplings(); 
  pars->setDependentParameters(); 
  pars->setDependentCouplings(); 
  // Set external particle masses for this matrix element
  mME.push_back(pars->ZERO); 
  mME.push_back(pars->ZERO); 
  mME.push_back(pars->mdl_MT); 
  mME.push_back(pars->mdl_MT); 
  static thrust::complex<double> tIPC[2] = {pars->GC_10, pars->GC_11}; 
  static double tIPD[2] = {pars->mdl_MT, pars->mdl_WT}; 

  hipMemcpyToSymbol(HIP_SYMBOL(cIPC), tIPC, 2 * sizeof(thrust::complex<double> )); 
  hipMemcpyToSymbol(HIP_SYMBOL(cIPD), tIPD, 2 * sizeof(double)); 
}

//--------------------------------------------------------------------------
// Evaluate |M|^2, part independent of incoming flavour.

__global__ void sigmaKin(hipPitchedPtr tp, double * meDevPtr, size_t mePitch) 
{
  // Set the parameters which change event by event
  // Need to discuss this with Stefan
  // pars->setDependentParameters();
  // pars->setDependentCouplings();

  // Reset color flows

  // for (int xx = 0; xx < 384; ++xx) {
  int nprocesses = 1; 
  int dim = blockIdx.x * blockDim.x + threadIdx.x; 

  char * devPtr = (char * )tp.ptr; 
  size_t dpt = tp.pitch; 
  size_t slicePitch = dpt * 4; 

  char * dps = devPtr + dim * slicePitch; 
  double * matrix_element = (double * )((char * )meDevPtr + dim * mePitch); 

  thrust::complex<double> amp[3]; 


  // Local variables and constants
  const int ncomb = 16; 
  // static bool goodhel[ncomb] = {ncomb * false};
  // static int ntry = 0, sum_hel = 0, ngood = 0;
  // static int igood[ncomb];
  // static int jhel;
  // std::complex<double> **wfs;
  double t[1]; 
  // Helicities for the process
  // static const int helicities[ncomb][nexternal] =
  // {{-1,-1,-1,-1},{-1,-1,-1,1},{-1,-1,1,-1},{-1,-1,1,1},{-1,1,-1,-1},{-1,1,-1,
  // 1},{-1,1,1,-1},{-1,1,1,1},{1,-1,-1,-1},{1,-1,-1,1},{1,-1,1,-1},{1,-1,1,1},{
  // 1,1,-1,-1},{1,1,-1,1},{1,1,1,-1},{1,1,1,1}};
  // Denominators: spins, colors and identical particles
  const int denominators[1] = {256}; 


  // Reset the matrix elements
  for(int i = 0; i < nprocesses; i++ )
  {
    matrix_element[i] = 0.; 
  }
  // Define permutation
  // int perm[nexternal];
  // for(int i = 0; i < nexternal; i++){
  // perm[i]=i;
  // }


  for (int ihel = 0; ihel < ncomb; ihel++ )
  {
    calculate_wavefunctions(ihel, dps, dpt, matrix_element[0]); 
  }


  for (int i = 0; i < nprocesses; ++ i)
  {
    matrix_element[i] /= denominators[i]; 
  }


}

//==========================================================================
// Private class member functions

//--------------------------------------------------------------------------


